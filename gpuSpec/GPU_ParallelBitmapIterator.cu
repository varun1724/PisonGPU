#include "hip/hip_runtime.h"
#include "GPU_ParallelBitmapIterator.h"
#include <sys/time.h>

struct ParallelBitmapMetadata {
    int start_word_id;
    int end_word_id;
    unsigned long* quote_bitmap;
    unsigned long* lev_colon_bitmap[MAX_LEVEL + 1];
    unsigned long* lev_comma_bitmap[MAX_LEVEL + 1];
};

ParallelBitmapMetadata pb_metadata[MAX_THREAD];

typedef struct CommaPosInfo {
    int thread_id;
    int level;
    long start_pos;
    long end_pos;
    long* comma_positions;
    long top_comma_positions;
}CommaPosInfo;

CommaPosInfo comma_pos_info[MAX_THREAD];

int num_of_threads = 128;


__global__ void generateCommaPositionsInKernel(ParallelBitmapMetadata* pb_metadata, CommaPosInfo* comma_pos_info, long start_pos, long end_pos, int level, int num_of_threads) {
    comma_pos_info[threadIdx.x].level = level;
    comma_pos_info[threadIdx.x].start_pos = start_pos;
    comma_pos_info[threadIdx.x].end_pos = end_pos;

    comma_pos_info[threadIdx.x].comma_positions = new long[MAX_NUM_ELE / num_of_threads + 1];
    comma_pos_info[threadIdx.x].top_comma_positions = -1;

    unsigned long* levels = pb_metadata[threadIdx.x].lev_comma_bitmap[level];
    if (levels == NULL) {
        return;
    }

    unsigned long commabit;
    long cur_start_pos = pb_metadata[threadIdx.x].start_word_id;
    long cur_end_pos = pb_metadata[threadIdx.x].end_word_id;
    long st = cur_start_pos > (start_pos / 64) ? cur_start_pos : (start_pos / 64);
    long ed = cur_end_pos < (ceil(double(end_pos) / 64)) ? cur_end_pos : (ceil(double(end_pos) / 64));
    for (long i = st; i < ed; ++i) {
        unsigned long idx = 0;
        if (threadIdx.x >= 1) idx = i - cur_start_pos;
        else idx = i;
        commabit = levels[idx];
        while (commabit) {

            // Counts trailing zeroes
            int zeroCount = 0;
            if (commabit == 0) {
                zeroCount = 64; // All bits are zeros, so there are 64 trailing zeros
            } else {
                while ((commabit & 1) == 0) {
                    commabit >>= 1;
                    zeroCount++;
                }
            }

            long offset = i * 64 + zeroCount;
            if (start_pos <= offset && offset <= end_pos) {
                comma_pos_info[threadIdx.x].comma_positions[++comma_pos_info[threadIdx.x].top_comma_positions] = offset;
            }
            commabit = commabit & (commabit - 1);
        }
    }
}


void GPUParallelBitmapIterator::generateCommaPositionsParallel(long start_pos, long end_pos, int level, long* comma_positions, long& top_comma_positions) {
    int start_chunk = -1;
    int end_chunk = -1;
    int chunk_num = mGPUParallelBitmap->mThreadNum;
    for (int i = mCurChunkId; i < chunk_num; ++i) {
        if (pb_metadata[i].start_word_id <= (start_pos / 64)) {
            start_chunk = i;
        }
        if (pb_metadata[i].end_word_id >= (ceil(double(end_pos) / 64)) && end_chunk == -1) {
            end_chunk = i;
        }
        if (start_chunk > -1 && end_chunk > -1) break;
    }
    if(start_chunk == 0 && end_chunk == -1) end_chunk = 0;
    mCurChunkId = start_chunk;

    if (start_chunk == end_chunk) {
        cout << "No threading occurred" << endl;
        return;
    }

    CommaPosInfo* cuda_comma_pos_info;
    ParallelBitmapMetadata* cuda_pb_metadata;

    hipMalloc(&cuda_comma_pos_info, sizeof(comma_pos_info));
    hipMalloc(&cuda_pb_metadata, sizeof(pb_metadata));

    hipMemcpy(cuda_comma_pos_info, comma_pos_info, sizeof(comma_pos_info), hipMemcpyHostToDevice);
    hipMemcpy(cuda_pb_metadata, pb_metadata, sizeof(pb_metadata), hipMemcpyHostToDevice);

    generateCommaPositionsInKernel<<<1, end_chunk>>>(cuda_pb_metadata, cuda_comma_pos_info, start_pos, end_pos, level, num_of_threads);

    hipMemcpy(cuda_comma_pos_info, comma_pos_info, sizeof(comma_pos_info), hipMemcpyDeviceToHost);
    hipFree(cuda_comma_pos_info);

    for (int i = start_chunk; i <= end_chunk; ++i) {
        for (int j = 0; j <= comma_pos_info[i].top_comma_positions; ++j) {
            comma_positions[++top_comma_positions] = comma_pos_info[i].comma_positions[j];
        }
        free(comma_pos_info[i].comma_positions);
    }

}


// Saving metadata of linked leveled bitmap in consecutive order can further improve the performance.
void GPUParallelBitmapIterator::gatherParallelBitmapInfo() {
    int chunk_num = mGPUParallelBitmap->mThreadNum;
    int depth = mGPUParallelBitmap->mDepth;
    for (int chunk_id = 0; chunk_id < chunk_num; ++chunk_id) {
        pb_metadata[chunk_id].start_word_id = mGPUParallelBitmap->mBitmaps[chunk_id]->mStartWordId;
        pb_metadata[chunk_id].end_word_id = mGPUParallelBitmap->mBitmaps[chunk_id]->mEndWordId;
        pb_metadata[chunk_id].quote_bitmap = mGPUParallelBitmap->mBitmaps[chunk_id]->mQuoteBitmap;
        for (int l = 0; l <= depth; ++l) {
            pb_metadata[chunk_id].lev_colon_bitmap[l] = mGPUParallelBitmap->mBitmaps[chunk_id]->mFinalLevColonBitmap[l];
            pb_metadata[chunk_id].lev_comma_bitmap[l] = mGPUParallelBitmap->mBitmaps[chunk_id]->mFinalLevCommaBitmap[l];
        }
    }
}

void GPUParallelBitmapIterator::generateColonPositions(long start_pos, long end_pos, int level, long* colon_positions, long& top_colon_positions) {
    // find starting and ending chunks in linked leveled colon bitmaps
    int start_chunk = -1;
    int end_chunk = -1;
    int thread_num = mGPUParallelBitmap->mThreadNum;
    for (int i = mCurChunkId; i < thread_num; ++i) {
        if (pb_metadata[i].start_word_id <= (start_pos / 64)) {
            start_chunk = i;
        }
        if (pb_metadata[i].end_word_id >= (ceil(double(end_pos) / 64)) && end_chunk == -1) {
            end_chunk = i;
        }
        if (start_chunk > -1 && end_chunk > -1) break;
    }
    if(start_chunk == 0 && end_chunk == -1) end_chunk = 0;
    mCurChunkId = start_chunk;
    // iterate through the corresponding linked leveled colon bitmaps
    int cur_chunk = start_chunk;
    while (cur_chunk <= end_chunk) {
        unsigned long* levels = pb_metadata[cur_chunk].lev_colon_bitmap[level];
        if (levels == NULL) {
            ++cur_chunk;
            continue;
        }
        unsigned long colonbit;
        long cur_start_pos = pb_metadata[cur_chunk].start_word_id;
        long cur_end_pos = pb_metadata[cur_chunk].end_word_id;
        long st = cur_start_pos > (start_pos / 64) ? cur_start_pos : (start_pos / 64);
        long ed = cur_end_pos < (ceil(double(end_pos) / 64)) ? cur_end_pos : (ceil(double(end_pos) / 64));
        for (long i = st; i < ed; ++i) {
            unsigned long idx = 0;
            if (cur_chunk >= 1) idx = i - cur_start_pos;
            else idx = i;
            colonbit = levels[idx];
            int cnt = __builtin_popcountl(colonbit);
            while (colonbit) {
                long offset = i * 64 + __builtin_ctzll(colonbit);
                if (start_pos <= offset && offset <= end_pos) {
                    colon_positions[++top_colon_positions] = offset;
                }
                colonbit = colonbit & (colonbit - 1);
            }
        }
        ++cur_chunk;
    }
}

void GPUParallelBitmapIterator::generateCommaPositions(long start_pos, long end_pos, int level, long* comma_positions, long& top_comma_positions) {
    // find starting and ending chunks in linked leveled comma bitmaps
    int start_chunk = -1;
    int end_chunk = -1;
    int chunk_num = mGPUParallelBitmap->mThreadNum;
    for (int i = mCurChunkId; i < chunk_num; ++i) {
        if (pb_metadata[i].start_word_id <= (start_pos / 64)) {
            start_chunk = i;
        }
        if (pb_metadata[i].end_word_id >= (ceil(double(end_pos) / 64)) && end_chunk == -1) {
            end_chunk = i;
        }
        if (start_chunk > -1 && end_chunk > -1) break;
    }
    if(start_chunk == 0 && end_chunk == -1) end_chunk = 0;
    mCurChunkId = start_chunk;
    // iterate through the corresponding linked leveled comma bitmaps
    int cur_chunk = start_chunk;
    while (cur_chunk <= end_chunk) {
        unsigned long* levels = pb_metadata[cur_chunk].lev_comma_bitmap[level];
        if (levels == NULL) {
            ++cur_chunk;
            continue;
        }
        unsigned long commabit;
        long cur_start_pos = pb_metadata[cur_chunk].start_word_id;
        long cur_end_pos = pb_metadata[cur_chunk].end_word_id;
        long st = cur_start_pos > (start_pos / 64) ? cur_start_pos : (start_pos / 64);
        long ed = cur_end_pos < (ceil(double(end_pos) / 64)) ? cur_end_pos : (ceil(double(end_pos) / 64));
        for (long i = st; i < ed; ++i) {
            unsigned long idx = 0;
            if (cur_chunk >= 1) idx = i - cur_start_pos;
            else idx = i;
            commabit = levels[idx];
            int cnt = __builtin_popcountl(commabit);
            while (commabit) {
                long offset = i * 64 + __builtin_ctzll(commabit);
                if (start_pos <= offset && offset <= end_pos) {
                    comma_positions[++top_comma_positions] = offset;
                }
                commabit = commabit & (commabit - 1);
            }
        }
        ++cur_chunk;
    }
}

bool GPUParallelBitmapIterator::findFieldQuotePos(long colon_pos, long& start_pos, long& end_pos) {
    long w_id = colon_pos/64;
    long offset = colon_pos%64;
    long start_quote = 0;
    long end_quote = 0;
    start_pos = 0; end_pos = 0;
    int cur_chunk = -1;
    int chunk_num = mGPUParallelBitmap->mThreadNum;
    // find the chunk where the current colon is in
    for (int i = mCurChunkId; i < chunk_num; ++i) {
        if (w_id >= pb_metadata[i].start_word_id && w_id < pb_metadata[i].end_word_id) {
            cur_chunk = i;
            break;
        }
    }
    if (cur_chunk == -1) {
        return false;
    }
    while (w_id >= 0)
    {
        // check whether the current chunk needs to be updated
        if (w_id < pb_metadata[cur_chunk].start_word_id) {
            //cout<<"update chunk id "<<cur_chunk<<endl;
            if ((--cur_chunk) == -1) {
                return false;
            }
        }
        long quote_id = w_id - pb_metadata[cur_chunk].start_word_id;
        unsigned long quotebit = pb_metadata[cur_chunk].quote_bitmap[quote_id];
        unsigned long offset = w_id * 64 + __builtin_ctzll(quotebit);
        while (quotebit && offset < colon_pos)
        {
            if (end_pos != 0)
            {
                start_quote = offset;
            }
            else if(start_quote == 0)
            {
                start_quote = offset;
            }
            else if(end_quote == 0)
            {
                end_quote = offset;
            }
            else
            {
                start_quote = end_quote;
                end_quote = offset;
            }
            quotebit = quotebit & (quotebit - 1);
            offset = w_id * 64 + __builtin_ctzll(quotebit);
        }
        if(start_quote != 0 && end_quote == 0)
        {
            end_quote = start_quote;
            start_quote = 0;
            end_pos = end_quote;
        }
        else if(start_quote != 0 && end_quote != 0)
        {
            start_pos = start_quote;
            end_pos = end_quote;
            return true;
        }
        --w_id;
    }
    return false;
}

GPUParallelBitmapIterator* GPUParallelBitmapIterator::getCopy() {
    GPUParallelBitmapIterator* pbi = new GPUParallelBitmapIterator();
    pbi->mGPUParallelBitmap = mGPUParallelBitmap;
    pbi->mCurLevel = mCurLevel;
    pbi->mTopLevel = mCurLevel;
    pbi->mCurChunkId = mCurChunkId;
    pbi->mFindDomArray = mFindDomArray;
    if (pbi->mTopLevel >= 0) {
        pbi->mCtxInfo[mCurLevel].type = mCtxInfo[mCurLevel].type;
        pbi->mCtxInfo[mCurLevel].positions = mCtxInfo[mCurLevel].positions;
        pbi->mCtxInfo[mCurLevel].start_idx = mCtxInfo[mCurLevel].start_idx;
        pbi->mCtxInfo[mCurLevel].end_idx = mCtxInfo[mCurLevel].end_idx;
        pbi->mCtxInfo[mCurLevel].cur_idx = -1;
        pbi->mPosArrAlloc[mCurLevel] = mPosArrAlloc[mCurLevel];
        pbi->mCtxInfo[mCurLevel + 1].positions = NULL;
        for (int i = mCurLevel + 1; i < MAX_LEVEL; ++i) {
            pbi->mPosArrAlloc[i] = false;
            pbi->mPosArrAlloc[i] = NULL;
        }
    }
    pbi->mCopiedIterator = true;
    return pbi;
}

bool GPUParallelBitmapIterator::up() {
    if (mCurLevel == mTopLevel) return false;
    --mCurLevel;
    return true;
}

bool GPUParallelBitmapIterator::down() {
    if (mCurLevel < mTopLevel || mCurLevel > mGPUParallelBitmap->mDepth) return false;
    ++mCurLevel;
    long  start_pos = -1;
    long end_pos = -1;
    int thread_num = mGPUParallelBitmap->mThreadNum;
    if (mCurLevel == mTopLevel + 1) {
        if (mTopLevel == -1) {
            long text_length = mGPUParallelBitmap->mRecordLength;
            start_pos = 0;
            end_pos = text_length;
            mCtxInfo[mCurLevel].positions = (long*)malloc((text_length / thread_num + 1) * sizeof (long));
            mPosArrAlloc[mCurLevel] = true;
        } else {
            long cur_idx = mCtxInfo[mCurLevel - 1].cur_idx;
            start_pos = mCtxInfo[mCurLevel - 1].positions[cur_idx];
            end_pos = mCtxInfo[mCurLevel - 1].positions[cur_idx + 1];
            if (mCtxInfo[mCurLevel].positions == NULL || mPosArrAlloc[mCurLevel] == false) {
                mCtxInfo[mCurLevel].positions = (long*)malloc((MAX_NUM_ELE / thread_num + 1) * sizeof (long));
                mPosArrAlloc[mCurLevel] = true;
            }
        }
        mCtxInfo[mCurLevel].start_idx = 0;
        mCtxInfo[mCurLevel].cur_idx = -1;
        mCtxInfo[mCurLevel].end_idx = -1;
    } else {
        long cur_idx = mCtxInfo[mCurLevel - 1].cur_idx;
        if (cur_idx > mCtxInfo[mCurLevel - 1].end_idx) {
            --mCurLevel;
            return false;
        }
        start_pos = mCtxInfo[mCurLevel - 1].positions[cur_idx];
        end_pos = mCtxInfo[mCurLevel - 1].positions[cur_idx + 1];
        mCtxInfo[mCurLevel].positions = mCtxInfo[mCurLevel - 1].positions;
        mCtxInfo[mCurLevel].start_idx = mCtxInfo[mCurLevel - 1].end_idx + 1;
        mCtxInfo[mCurLevel].cur_idx = mCtxInfo[mCurLevel - 1].end_idx;
        mCtxInfo[mCurLevel].end_idx = mCtxInfo[mCurLevel - 1].end_idx;
    }
    long i = start_pos;
    if (start_pos > 0 || mCurLevel > 0) ++i;
    char ch = mGPUParallelBitmap->mRecord[i];
    while (i < end_pos && (ch == ' ' || ch == '\n')) {
        ch = mGPUParallelBitmap->mRecord[++i];
    }
    if (mGPUParallelBitmap->mRecord[i] == '{') {
        mCtxInfo[mCurLevel].type = OBJECT;
        generateColonPositions(i, end_pos, mCurLevel, mCtxInfo[mCurLevel].positions, mCtxInfo[mCurLevel].end_idx);
        return true;
    } else if (mGPUParallelBitmap->mRecord[i] == '[') {
        mCtxInfo[mCurLevel].type = ARRAY;
        if (mFindDomArray == false && (end_pos - i + 1) > SINGLE_THREAD_MAX_ARRAY_SIZE) {
            generateCommaPositionsParallel(i, end_pos, mCurLevel, mCtxInfo[mCurLevel].positions, mCtxInfo[mCurLevel].end_idx);
            mFindDomArray = true;
        } else {
            generateCommaPositions(i, end_pos, mCurLevel, mCtxInfo[mCurLevel].positions, mCtxInfo[mCurLevel].end_idx);
        }
        return true;
    }
    --mCurLevel;
    return false;
}

bool GPUParallelBitmapIterator::isObject() {
    if (mCurLevel >= 0 && mCurLevel <= mGPUParallelBitmap->mDepth && mCtxInfo[mCurLevel].type == OBJECT) {
        return true;
    }
    return false;
}

bool GPUParallelBitmapIterator::isArray() {
    if (mCurLevel >= 0 && mCurLevel <= mGPUParallelBitmap->mDepth && mCtxInfo[mCurLevel].type == ARRAY) {
        return true;
    }
    return false;
}

bool GPUParallelBitmapIterator::moveNext() {
    if (mCurLevel < 0 || mCurLevel > mGPUParallelBitmap->mDepth || mCtxInfo[mCurLevel].type != ARRAY) return false;
    long next_idx = mCtxInfo[mCurLevel].cur_idx + 1;
    if (next_idx >= mCtxInfo[mCurLevel].end_idx) return false;
    mCtxInfo[mCurLevel].cur_idx = next_idx;
    return true;
}

bool GPUParallelBitmapIterator::moveToKey(char* key) {
    if (mCurLevel < 0 || mCurLevel > mGPUParallelBitmap->mDepth || mCtxInfo[mCurLevel].type != OBJECT) return false;
    long cur_idx = mCtxInfo[mCurLevel].cur_idx + 1;
    long end_idx = mCtxInfo[mCurLevel].end_idx;
    while (cur_idx < end_idx) {
        long colon_pos = mCtxInfo[mCurLevel].positions[cur_idx];
        long start_pos = 0, end_pos = 0;
        if (!findFieldQuotePos(colon_pos, start_pos, end_pos)) {
            return false;
        }
        int key_size = end_pos - start_pos - 1;
        if (key_size == strlen(key)) {
            memcpy(mKey, mGPUParallelBitmap->mRecord + start_pos + 1, key_size);
            mKey[end_pos - start_pos - 1] = '\0';
            if (memcmp(mKey, key, key_size) == 0) {
                mCtxInfo[mCurLevel].cur_idx = cur_idx;
                return true;
            }
        }
        ++cur_idx;
    }
    return false;
}

char* GPUParallelBitmapIterator::moveToKey(unordered_set<char*>& key_set) {
    if (key_set.empty() == true || mCurLevel < 0 || mCurLevel > mGPUParallelBitmap->mDepth || mCtxInfo[mCurLevel].type != OBJECT) return NULL;
    long cur_idx = mCtxInfo[mCurLevel].cur_idx + 1;
    long end_idx = mCtxInfo[mCurLevel].end_idx;
    while (cur_idx < end_idx) {
        long colon_pos = mCtxInfo[mCurLevel].positions[cur_idx];
        long start_pos = 0, end_pos = 0;
        if (!findFieldQuotePos(colon_pos, start_pos, end_pos)) {
            return NULL;
        }
        bool has_m_key = false;
        unordered_set<char*>::iterator iter;
        for (iter = key_set.begin(); iter != key_set.end(); ++iter) {
            char* key = (*iter);
            int key_size = end_pos - start_pos - 1;
            if (key_size == strlen(key)) {
                if (has_m_key == false) {
                    memcpy(mKey, mGPUParallelBitmap->mRecord + start_pos + 1, key_size);
                    mKey[end_pos - start_pos - 1] = '\0';
                    has_m_key = true;
                }
                if (memcmp(mKey, key, key_size) == 0) {
                    mCtxInfo[mCurLevel].cur_idx = cur_idx;
                    key_set.erase(iter);
                    return key;
                }
            }
        }
        ++cur_idx;
    }
    mCtxInfo[mCurLevel].cur_idx = cur_idx;
    return NULL;
}

int GPUParallelBitmapIterator::numArrayElements() {
    if (mCurLevel >= 0 && mCurLevel <= mGPUParallelBitmap->mDepth && mCtxInfo[mCurLevel].type == ARRAY) {
        return mCtxInfo[mCurLevel].end_idx - mCtxInfo[mCurLevel].start_idx;
    }
    return 0;
}

bool GPUParallelBitmapIterator::moveToIndex(int index) {
    if (mCurLevel < 0 || mCurLevel > mGPUParallelBitmap->mDepth || mCtxInfo[mCurLevel].type != ARRAY) return false;
    long next_idx = mCtxInfo[mCurLevel].start_idx + index;
    if (next_idx > mCtxInfo[mCurLevel].end_idx) return false;
    mCtxInfo[mCurLevel].cur_idx = next_idx;
    return true;
}

char* GPUParallelBitmapIterator::getValue() {
    if (mCurLevel < 0 || mCurLevel > mGPUParallelBitmap->mDepth) return NULL;
    long cur_idx = mCtxInfo[mCurLevel].cur_idx;
    long next_idx = cur_idx + 1;
    if (next_idx > mCtxInfo[mCurLevel].end_idx) return NULL;
    // current ':' or ','
    long cur_pos = mCtxInfo[mCurLevel].positions[cur_idx];
    // next ':' or ','
    long next_pos = mCtxInfo[mCurLevel].positions[next_idx];
    int type = mCtxInfo[mCurLevel].type;
    if (type == OBJECT && next_idx < mCtxInfo[mCurLevel].end_idx) {
        long start_pos = 0, end_pos = 0;
        if (findFieldQuotePos(next_pos, start_pos, end_pos) == false) {
            return "";
        }
        // next quote
        next_pos = start_pos;
    }
    long text_length = next_pos - cur_pos - 1;
    if (text_length <= 0) return "";
    char* ret = (char*)malloc(text_length + 1);
    memcpy(ret, mGPUParallelBitmap->mRecord + cur_pos + 1, text_length);
    ret[text_length] = '\0';
    return ret;
}
